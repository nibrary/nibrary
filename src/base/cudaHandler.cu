#include "config.h"

#ifdef HAVE_CUDA

#include "cudaHandler.cuh"

using namespace NIBR;

bool                NIBR::CUDAHANDLER::cuda_initialized = false;
hipDeviceProp_t      NIBR::CUDAHANDLER::cuda_dev_prop;
int                 NIBR::CUDAHANDLER::cuda_maxThreadsPerBlock;
int                 NIBR::CUDAHANDLER::cuda_maxThreadsDim[3];
int                 NIBR::CUDAHANDLER::cuda_maxGridSize[3];

void NIBR::CUDAHANDLER::CUDA_PRINT_ERROR(hipError_t cudaFunction)
{
    if (cudaFunction != hipSuccess){
         disp(MSG_ERROR,"%s",hipGetErrorString(cudaFunction));
    }
}

void NIBR::CUDAHANDLER::CUDA_INIT() 
{
    if (cuda_initialized==false) {
        hipDeviceReset();
        int devNo;
        hipGetDevice(&devNo);
        CUDA_PRINT_ERROR(hipGetDeviceProperties (&cuda_dev_prop, devNo)); 

        cuda_maxThreadsPerBlock = cuda_dev_prop.maxThreadsPerBlock;
        cuda_maxThreadsDim[0]   = cuda_dev_prop.maxThreadsDim[0];
        cuda_maxThreadsDim[1]   = cuda_dev_prop.maxThreadsDim[1];
        cuda_maxThreadsDim[2]   = cuda_dev_prop.maxThreadsDim[2];

        cuda_maxGridSize[0]     = cuda_dev_prop.maxGridSize[0];
        cuda_maxGridSize[1]     = cuda_dev_prop.maxGridSize[1];
        cuda_maxGridSize[2]     = cuda_dev_prop.maxGridSize[2];

        cuda_initialized = true;
    }
}

void NIBR::CUDAHANDLER::CUDA_EXIT() 
{
    hipDeviceReset();
}

void NIBR::CUDAHANDLER::CUDA_PRINT_INFO() 
{
    if (NIBR::VERBOSE()<VERBOSE_INFO)
        return;

    int devCount, devNo;
    hipGetDeviceCount(&devCount);
    hipGetDevice(&devNo);

    disp(MSG_INFO,"CUDA INFO");
    std::cout << "\033[32m";
    std::cout << "Device count:                 " << devCount << " (Using device #" << devNo << ")" <<std::endl;
    std::cout << "name (major,minor):           " << cuda_dev_prop.name << " (" << cuda_dev_prop.major << "." << cuda_dev_prop.minor << ")" << std::endl;
    std::cout << "computeMode:                  " << cuda_dev_prop.computeMode << std::endl;
    std::cout << "totalGlobalMem:               " << cuda_dev_prop.totalGlobalMem/1024/1024/1024 << " GB"<< std::endl;
    std::cout << "sharedMemPerBlock:            " << cuda_dev_prop.sharedMemPerBlock/1024 << " KB"<< std::endl;
    std::cout << "regsPerBlock:                 " << cuda_dev_prop.regsPerBlock << " 32-bit registers"<< std::endl;
    std::cout << "maxThreadsPerBlock:           " << cuda_maxThreadsPerBlock << std::endl;
    std::cout << "maxThreadsDim:                " << "[" << cuda_maxThreadsDim[0] << "," << cuda_maxThreadsDim[1] << "," << cuda_maxThreadsDim[2] << "]"<< std::endl;
    std::cout << "maxGridSize:                  " << "[" << cuda_maxGridSize[0] << "," << cuda_maxGridSize[1] << "," << cuda_maxGridSize[2] << "]"<< std::endl;
    std::cout << "multiProcessorCount:          " << cuda_dev_prop.multiProcessorCount << std::endl;
    std::cout << "maxThreadsPerMultiProcessor:  " << cuda_dev_prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "\033[0m";
    

}

#endif